#include "hip/hip_runtime.h"
#include <ctime>
#include <iomanip>
#include <iostream>

// success after add this comment -- should be VS issue.
// #pragma comment(lib,"cuda.lib")

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

// #include "hip/hip_vector_types.h"
#include "utils.h"
#include "utilsm.h"

#define MAX_THREADS 1024 * 1024 * 64
// #define MAX_THREADS 32 * 3072
// #define MAX_THREADS 64 * 1024

// #pragma comment(lib,"cudart.lib")
void generate_input(int *input, unsigned int input_len) {
    // std::srand(std::time(nullptr));
    for (unsigned int i = 0; i < input_len; ++i) {
        // input[i] = std::rand(0,VECTOR_LENGTH);
        input[i] = i;
    }
}

// golden test
void cpu_vector_add(int *vector1, int *const vector2, int *vector_sum,
                    unsigned int len) {
    for (unsigned int i = 0; i < len; ++i) {
        vector_sum[i] = vector1[i] + vector2[i];
        // vector1[i] += vector2[i];
    }
}

/**
 * @brief
 */
__global__ void gpu_vector_add(int *vector1, int *const vector2,
                               int *vector_sum, const unsigned int length) {
    // unsigned int tid;
    // for (unsigned int i = 0; i < length;) {
    //     tid = threadIdx.x + blockDim.x * blockIdx.x + i;
    //     i += MAX_THREADS;
    //     if ((tid < length) && (tid < i)) {
    //         vector_sum[tid] = vector1[tid] + vector2[tid];
    //         // vector1[tid] += vector2[tid];
    //     }
    // }
    // single cycle version
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < length) {
        vector_sum[tid] = vector1[tid] + vector2[tid];
        // vector1[tid] += vector2[tid];
    }
}

__global__ void gpu_vector_add4(int4 *vector1, int4 *const vector2,
                                int4 *vector_sum, const unsigned int length) {
    // unsigned int tid;
    // for (unsigned int i = 0; i < length;) {
    //     tid = threadIdx.x + blockDim.x * blockIdx.x + i;
    //     i += MAX_THREADS;
    //     if ((tid < length) && (tid < i)) {
    //         vector_sum[tid].x = vector1[tid].x + vector2[tid].x;
    //         vector_sum[tid].y = vector1[tid].y + vector2[tid].y;
    //         vector_sum[tid].z = vector1[tid].z + vector2[tid].z;
    //         vector_sum[tid].w = vector1[tid].w + vector2[tid].w;
    //     }
    // }
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < length) {
        vector_sum[tid].x = vector1[tid].x + vector2[tid].x;
        vector_sum[tid].y = vector1[tid].y + vector2[tid].y;
        vector_sum[tid].z = vector1[tid].z + vector2[tid].z;
        vector_sum[tid].w = vector1[tid].w + vector2[tid].w;
    }
}

int main_old() {
    // Set up clock for timing comparisons
    std::clock_t start;
    float duration_cpu, duration_gpu;
    unsigned int len_pow = 24;
    // std::cout << "CLOCKS_PER_SEC is " << CLOCKS_PER_SEC << std::endl;//10^6
    // for (; len_pow < 25; len_pow++) {
    std::cout << "---------------- len_pow is " << len_pow
              << " ----------------" << std::endl;
    unsigned int vector_len = (unsigned int)1 << len_pow;
    unsigned int length_bytes = vector_len * sizeof(int);
    std::cout << "vector length is " << vector_len << std::endl;
    std::cout << "vector size is "
              << static_cast<double>(vector_len / 1024) / 1024 * sizeof(int)
              << " GB." << std::endl;

    int *vector1 = new int[vector_len]; // on host
    int *vector2 = new int[vector_len];
    int *vector_sum = new int[vector_len];
    int *vector_sum2 = new int[vector_len];
    int *vector1_g, *vector2_g, *vector_sum_g; // in GPU

    // generate input
    generate_input(vector1, vector_len);
    generate_input(vector2, vector_len);
    start = std::clock();
    cpu_vector_add(vector1, vector2, vector_sum, vector_len);
    duration_cpu = std::clock() - start;
    std::cout << "CPU computation:" << std::endl;
    std::cout << "  duration: ";
    print_time_us(duration_cpu);

    // Set up device-side memory for input
    checkCudaErrors(hipMalloc(&vector1_g, length_bytes));
    checkCudaErrors(hipMalloc(&vector2_g, length_bytes));
    checkCudaErrors(hipMalloc(&vector_sum_g, length_bytes));
    checkCudaErrors(
        hipMemcpy(vector1_g, vector1, length_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(vector2_g, vector2, length_bytes, hipMemcpyHostToDevice));

    // generate CUDA parameters and call CUDA device functions
    // unsigned int max_threads_used;
    // if (vector_len > MAX_THREADS) {
    //     max_threads_used = MAX_THREADS;
    //     std::cout << "MAX_THREADS is: " << MAX_THREADS << std::endl
    //               << "while vector_len is: " << vector_len << std::endl;
    // } else {
    //     max_threads_used = vector_len;
    // }
    // unsigned int block_sz = 1024;
    // unsigned int grid_sz = max_threads_used / block_sz;
    // if (max_threads_used % block_sz) grid_sz++;
    // hipEvent_t start_g, stop_g;
    // checkCudaErrors(hipEventCreate(&start_g));
    // checkCudaErrors(hipEventCreate(&stop_g));
    // checkCudaErrors(hipEventRecord(start_g));
    // gpu_vector_add<<<grid_sz, block_sz>>>(vector1_g, vector2_g, vector_sum_g,
    //                                       vector_len);
    // checkCudaErrors(hipEventRecord(stop_g));
    // checkCudaErrors(hipEventSynchronize(stop_g));
    // checkCudaErrors(hipEventElapsedTime(&duration_gpu, start_g, stop_g));
    // std::cout << "GPU computation:" << std::endl;
    // std::cout << "  duration: " << duration_gpu << "ms" << std::endl;
    // checkCudaErrors(hipEventDestroy(start_g));
    // checkCudaErrors(hipEventDestroy(stop_g));

    // checkCudaErrors(hipMemcpy(vector_sum2, vector_sum_g, length_bytes,
    //                            hipMemcpyDeviceToHost));
    // // checkCudaErrors(hipMemcpy(vector_sum2, vector1_g, length_bytes,
    // //                            hipMemcpyDeviceToHost));

    // int4 version
    unsigned int max_threads_used = vector_len / 4;
    if (max_threads_used > MAX_THREADS) {
        max_threads_used = MAX_THREADS;
        std::cout << "MAX_THREADS is: " << MAX_THREADS << std::endl
                  << "while vector_len is: " << vector_len << std::endl;
    }
    unsigned int block_sz = 1024;
    unsigned int grid_sz = (max_threads_used + block_sz - 1) / block_sz;
    hipEvent_t start_g, stop_g;
    checkCudaErrors(hipEventCreate(&start_g));
    checkCudaErrors(hipEventCreate(&stop_g));
    checkCudaErrors(hipEventRecord(start_g));
    gpu_vector_add4<<<grid_sz, block_sz>>>((int4 *)vector1_g, (int4 *)vector2_g,
                                           (int4 *)vector_sum_g,
                                           vector_len / 4);
    checkCudaErrors(hipEventRecord(stop_g));
    checkCudaErrors(hipEventSynchronize(stop_g));
    checkCudaErrors(hipEventElapsedTime(&duration_gpu, start_g, stop_g));
    std::cout << "GPU computation:" << std::endl;
    std::cout << "  duration: " << duration_gpu << "ms" << std::endl;
    checkCudaErrors(hipEventDestroy(start_g));
    checkCudaErrors(hipEventDestroy(stop_g));
    checkCudaErrors(hipMemcpy(vector_sum2, vector_sum_g, length_bytes,
                               hipMemcpyDeviceToHost));

    // acceleration rate
    std::cout << "Calculation Result -- ";
    bool result = true;
    for (unsigned int i = 0; i < vector_len; i++) {
        if (vector_sum[i] != vector_sum2[i]) {
            std::cout << "Data mismatch at " << i << ": cpu_sum is "
                      << vector_sum[i] << " while gpu sum is " << vector_sum2[i]
                      << std::endl;
            result = false;
            print_vector(vector_sum, 0, 16);
            print_vector(vector_sum, vector_len - 16, vector_len);
            print_vector(vector_sum2, 0, 16);
            print_vector(vector_sum2, vector_len - 16, vector_len);
            break;
        }
    }
    std::cout << std::boolalpha << result << std::endl;
    std::cout << "Acceleration Rate: " << std::resetiosflags(std::ios::fixed)
              << duration_cpu / 1000 / duration_gpu << " times" << std::endl;
    std::cout << std::endl;

    delete[] vector1;
    delete[] vector2;
    delete[] vector_sum;
    delete[] vector_sum2;
    // }
    return 0;
}
