#include "hip/hip_runtime.h"
#include "reduce.h"

#include <cmath>
#include <iomanip>

#include "hip/hip_runtime.h"
#include ""

#include "utils.h"
#include "utilsm.h"

/**
 * @brief rough implementation
 *				all data W/R operation are on global memory
 * @param d_in 
 * @param d_in_len 
 * @return  
 */
__global__ void reduce_base(unsigned int* d_in, unsigned short d_in_len){
	d_in_len = d_in_len / 2;
  unsigned int tid;
	while(d_in_len > 0){
    tid = threadIdx.x + blockDim.x * blockIdx.x;
		if (tid < d_in_len){
			d_in[tid] += d_in[tid + d_in_len];
		}
		__syncthreads();
		d_in_len = d_in_len / 2;
	}
}

/**
 * \brief test reduce algorithm.
 * \param d_in 
 * \param d_in_len must be the power of 2. not a generate purpose function.
 *				should <= MAX_BLOCK_SZ
 * \return 
 */
unsigned int gpu_sum_reduce(unsigned int* hostMem, unsigned int vector_len,unsigned long long *time_total ){
	unsigned int* globalMem;
	unsigned long long time_start;
  unsigned long long duration_mmalc,duration_cp1, duration_cal, duration_cp2, duration_mfree;

	// copy data to device -- includes memory malloc time
	time_start = get_time_us();
	checkCudaErrors(hipMalloc(&globalMem, sizeof(unsigned int) * vector_len));
	duration_mmalc = get_time_us() - time_start;
	std::cout << "hipMalloc time is: ";
  print_time_us(duration_mmalc);
  time_start = get_time_us();
	checkCudaErrors(hipMemcpy(globalMem,hostMem , sizeof(unsigned int) * vector_len, 
		hipMemcpyHostToDevice));
	duration_cp1 = get_time_us() - time_start;
	bandwidth_print(duration_cp1, *hostMem, vector_len);
	// calculation
  unsigned int block_sz = vector_len / 2;
  unsigned int grid_sz = 1;
	if (vector_len > MAX_BLOCK_SZ) {
		block_sz = MAX_BLOCK_SZ;
		grid_sz = vector_len / MAX_BLOCK_SZ;
		if (vector_len % MAX_BLOCK_SZ) grid_sz += 1;
	}
	std::cout << "block_sz is " << block_sz << "  grid_sz is " << grid_sz << std::endl;
	time_start = get_time_us();
	reduce_base <<<grid_sz , block_sz, 2 * sizeof(unsigned int) * vector_len >>>
							(globalMem, vector_len);
	duration_cal = get_time_us() - time_start;
	std::cout << "  calculation duration is:";
  print_time_us(duration_cal);
  // copy back to host -- ignore due to we only need sum
	time_start = get_time_us();
	checkCudaErrors(hipMemcpy(hostMem, globalMem, sizeof(unsigned int),
		hipMemcpyDeviceToHost));
	duration_cp2 = get_time_us() - time_start;
  std::cout << "  Copy back " << std::endl;
  bandwidth_print(duration_cp2, *hostMem, sizeof(unsigned int)); 
	time_start = get_time_us();
  checkCudaErrors(hipFree(globalMem));
	duration_mfree = get_time_us() - time_start;
  std::cout << "  free time is: ";
  print_time_us(duration_mfree);
	// return globalMem[0];	// Must not use host memory rather than Device memory!!!
	*time_total = duration_mmalc + duration_cp1 + duration_cal + duration_cp2 + duration_mfree;
	return hostMem[0];
}

